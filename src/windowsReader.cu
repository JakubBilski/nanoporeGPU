#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <exception>
#include <time.h>

#include <hip/hip_runtime.h>

#include "defines.cuh"
#include "utils.cuh"
#include "graphCreationK.cuh"
#include "weakLeavesDeletionK.cuh"
#include "debugTools.cuh"
#include "tempOperations.cuh"

#include "defines.h"
#include "Correct.h"

template <int TNoBlocks> int* precleanedJumpGPU(std::fstream& fs, std::fstream& ts);

int main(int argc, char* argv[])
{
#ifdef DEBUG
	TestCorrect();
	return 0;
#else
    if(argc != 2 && argc != 3)
    {
      printf("Usage: reader file_path temp_file_path\n");
      return 0;
    }
	char* tempFilePath = "./tempFile.txt";
	if (argc == 3)
	{
		tempFilePath = argv[2];
	}
	char* inputFilePath = argv[1];
	printf("Machine:\n\t%d MB process memory\n", (sizeof(char)*HOST_CHUNK_SIZE) / (1024 * 1024));
	printf("\t%d MB device data memory\n", (sizeof(char)*DEVICE_CHUNK_SIZE) / (1024 * 1024));
	printf("\t%d MB device tree memory\n", (sizeof(int)*DEVICE_TREE_SIZE) / (1024 * 1024));
	printf("Starting\n");

	const int noTests = 1;
	for (size_t test = 0; test < noTests; test++)
	{
		printf(inputFilePath);
		printf(", %d-mers, run %d\n", MER_LENGHT, (int)test);

		std::fstream fs(inputFilePath, std::ios::in | std::ios::binary);
		assertOpenFile(fs, inputFilePath);
		std::fstream ts(tempFilePath, std::ios::in | std::ios::out | std::ios::binary | std::ofstream::trunc);
		assertOpenFile(ts, tempFilePath);
		clock_t start = clock();

		int* DBG = precleanedJumpGPU<10>(fs, ts);
		printf("%25s = %11f\n", "precleanedJumpGPU<10>", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();
		ts.close();

		char* tempFilePath2 = "result_file_aa.txt";
		fs.open(tempFilePath, std::ios::in | std::ios::binary);
		assertOpenFile(fs, tempFilePath);
		ts.open(tempFilePath2, std::ios::in | std::ios::out | std::ios::binary | std::ofstream::trunc);
		assertOpenFile(ts, tempFilePath2);
		start = clock();

		Correct(fs, ts, DBG);
		printf("%25s = %11f\n", "precleanedJumpGPU<10>", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();
		ts.close();

		printf("\n");
	}
	return 0;
#endif
}

template <int TNoBlocks>
int* precleanedJumpGPU(std::fstream& fs, std::fstream& ts)
{
	char* d_chunk;
	int* d_tree;
	int* d_treeLength;
	char* chunk = (char*)malloc(sizeof(char)*HOST_CHUNK_SIZE);
	char* clearedChunk = (char*)malloc(sizeof(char)*DEVICE_CHUNK_SIZE);
	int clearedChunkSize = 0;
	int chunkOffset = 0;
	int cutPhase = 2;
	gpuErrchk(hipMalloc(&d_chunk, DEVICE_CHUNK_SIZE * sizeof(char)));
	gpuErrchk(hipMalloc(&d_tree, DEVICE_TREE_SIZE * sizeof(int)));
	gpuErrchk(hipMemset(d_tree, 0, DEVICE_TREE_SIZE * sizeof(int)));
	gpuErrchk(hipMalloc(&d_treeLength, sizeof(int)));
	const int startingTreeLength = 4;
	gpuErrchk(hipMemcpy(d_treeLength, &startingTreeLength, sizeof(int), hipMemcpyHostToDevice));
	int lettersLength;
	while (!fs.eof())
	{
		fs.read(chunk + chunkOffset, HOST_CHUNK_SIZE - chunkOffset);
		int chunkLength = chunkOffset + fs.gcount();
		int i = 0;
		for (int j = 0; j < 3 - cutPhase; j++)
		{
			while (chunk[i] != '\n')
			{
				i++;
			}
			i++;
		}
		bool endOfChunk = false;
		while (!endOfChunk)
		{
			int startOfLetters = i;
			while (chunk[i] != '\n')
			{
				i++;
				if (clearedChunkSize + i - startOfLetters == DEVICE_CHUNK_SIZE)
				{
					gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
					AddPrecleanedChunkToGraph<MER_LENGHT> << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_tree, d_treeLength);
					AddPrecleanedChunkToTemp(ts, clearedChunk, clearedChunkSize);
					kernelErrchk();
					clearedChunkSize = 0;
				}
				if (i == chunkLength)
				{
					memcpy(chunk, chunk + startOfLetters, sizeof(char)*(chunkLength - startOfLetters));
					chunkOffset = chunkLength - startOfLetters;
					cutPhase = 3;
					endOfChunk = true;
					break;
				}
			}
			i++;
			if (!endOfChunk)
			{
				lettersLength = i - startOfLetters;
				memcpy(clearedChunk + clearedChunkSize, chunk + startOfLetters, lettersLength);
				clearedChunkSize += lettersLength;
				if (i + 2 >= chunkLength)
				{
					cutPhase = 0;
					break;
				}
				if (i + 2 + lettersLength >= chunkLength)
				{
					cutPhase = 1;
					break;
				}
				i += 2 + lettersLength;
				while (chunk[i] != '\n')
				{
					i++;
					if (i == chunkLength)
					{
						cutPhase = 2;
						endOfChunk = true;
						break;
					}
				}
				i++;
				if (i == chunkLength)
				{
					cutPhase = 3;
					chunkOffset = 0;
					break;
				}
			}
		}
	}
	gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
	AddPrecleanedChunkToGraph<MER_LENGHT> << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_tree, d_treeLength);
	kernelErrchk();
	DeleteWeakLeaves<MER_LENGHT> << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_tree);
	kernelErrchk();
	int finalTreeLength = 0;
	gpuErrchk(hipMemcpy(&finalTreeLength, d_treeLength, sizeof(int), hipMemcpyDeviceToHost));
	int* finalTree = (int*)malloc(sizeof(int)*finalTreeLength);
	gpuErrchk(hipMemcpy(finalTree, d_tree, finalTreeLength * sizeof(int), hipMemcpyDeviceToHost));
	//DisplaySizeInfo(finalTreeLength, MER_LENGHT);
	DisplayTree(finalTree);
	//DisplayTable(finalTree, finalTreeLength);
	gpuErrchk(hipFree(d_chunk));
	gpuErrchk(hipFree(d_tree));
	gpuErrchk(hipFree(d_treeLength));
	free(chunk);
	free(clearedChunk);
	return finalTree;
}