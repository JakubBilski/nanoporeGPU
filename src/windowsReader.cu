#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <exception>
#include <time.h>

#include <hip/hip_runtime.h>

#include "defines.cuh"
#include "utils.cuh"
#include "graphCreationK.cuh"

template <int TNoBlocks> void simpleGPU(std::ifstream& fs);
void stringCPU(std::ifstream& fs);
template <int TNoBlocks> void precleanedStreamGPU(std::ifstream& fs);
template <int TNoBlocks> void precleanedGPU(std::ifstream& fs);
void simpleCPU(std::ifstream& fs);

int main(int argc, char* argv[])
{
    if(argc != 2)
    {
      printf("Usage: reader file_path\n");
      return 0;
    }

	printf("Starting\n");

	const int noTests = 2;
	for (size_t test = 0; test < noTests; test++)
	{
		std::ifstream fs(argv[1], std::ios::in | std::ios::binary);
		clock_t start = clock();
		precleanedStreamGPU<1>(fs);
		printf("precleanedStreamGPU<1> done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();

		fs.open(argv[1], std::ios::in | std::ios::binary);
		start = clock();
		precleanedGPU<1>(fs);
		printf("precleanedGPU<1> done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();

		fs.open(argv[1], std::ios::in | std::ios::binary);
		start = clock();
		precleanedStreamGPU<5>(fs);
		printf("precleanedStreamGPU<5> done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();

		fs.open(argv[1], std::ios::in | std::ios::binary);
		start = clock();
		precleanedStreamGPU<10>(fs);
		printf("precleanedStreamGPU<10> done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();

		fs.open(argv[1], std::ios::in | std::ios::binary);
		start = clock();
		precleanedStreamGPU<20>(fs);
		printf("precleanedStreamGPU<20> done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();

		fs.open(argv[1], std::ios::in | std::ios::binary);
		start = clock();
		simpleCPU(fs);
		printf("simpleCPU done in %f seconds\n", 0.001f * (clock() - start) * 1000 / CLOCKS_PER_SEC);
		fs.close();
		printf("\n");
	}
	return 0;
}

template <int TNoBlocks>
void simpleGPU(std::ifstream& fs)
{
	char* d_chunk;
	int* d_out_numAs;
	int noAs = 0;
	char* chunk = (char*)malloc(sizeof(char)*INPUT_CHUNK_SIZE);
	gpuErrchk(hipMalloc(&d_chunk, INPUT_CHUNK_SIZE * sizeof(char)));
	gpuErrchk(hipMalloc(&d_out_numAs, sizeof(int)));
	gpuErrchk(hipMemset(&d_out_numAs, sizeof(int), 0));
	int chunkSize = 0;
	do
	{
		fs.read(chunk, INPUT_CHUNK_SIZE);
		chunkSize = fs.gcount();
		gpuErrchk(hipMemcpy(d_chunk, chunk, chunkSize * sizeof(char), hipMemcpyHostToDevice));
		AddChunkToGraph << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, chunkSize, d_out_numAs);
		kernelErrchk();
	} while (chunkSize == INPUT_CHUNK_SIZE);
	gpuErrchk(hipMemcpy(&noAs, d_out_numAs, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_chunk));
	gpuErrchk(hipFree(d_out_numAs));
	free(chunk);
	if (noAs != DEBUG_A_COUNT)
	{
		throw std::runtime_error("invalid value calculated by tested function");
	}
}

void stringCPU(std::ifstream& fs)
{
	std::string line;
	int noAs = 0;
	do
	{
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
		std::getline(fs, line);
		for (auto c : line)
		{
			if (c == 'A')
			{
				noAs++;
			}
		}
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	} while (!fs.eof());
	if (noAs != DEBUG_A_COUNT)
	{
		throw std::runtime_error("invalid value calculated by tested function");
	}
}

template <int TNoBlocks>
void precleanedStreamGPU(std::ifstream& fs)
{
	char* d_chunk;
	int* d_out_numAs;
	char* clearedChunk = (char*)malloc(sizeof(char)*INPUT_CHUNK_SIZE);
	int clearedChunkSize = 0;
	int noAs = 0;
	gpuErrchk(hipMalloc(&d_chunk, INPUT_CHUNK_SIZE * sizeof(char)));
	gpuErrchk(hipMalloc(&d_out_numAs, sizeof(int)));
	gpuErrchk(hipMemset(&d_out_numAs, sizeof(int), 0));
	while(!fs.eof())
	{
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
		int freeSpace = INPUT_CHUNK_SIZE - clearedChunkSize;
		fs.getline(&(clearedChunk[clearedChunkSize]), freeSpace);
		if (fs.fail())	//if line didn't fit into chunk
		{
			if (fs.eof())
			{
				break;
			}
			//printf("Kernel launch\n");
			gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
			AddPrecleanedChunkToGraph << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_out_numAs);
			kernelErrchk();
			int savedLen = fs.gcount() - 1;
			fs.clear();
			memcpy(clearedChunk, clearedChunk + clearedChunkSize, sizeof(char)*savedLen);
			clearedChunkSize = savedLen;
			fs.getline(&(clearedChunk[clearedChunkSize]), INPUT_CHUNK_SIZE);	
			if (fs.fail())
			{
				printf("Unhandled error\n");	//case when some line is longer than INPUT_CHUNK_SIZE
				exit(0);
			}
			clearedChunkSize += fs.gcount();
		}
		else
		{
			clearedChunkSize += fs.gcount();
		}
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
		fs.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	}
	//printf("Kernel launch\n");
	gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
	AddPrecleanedChunkToGraph << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_out_numAs);
	kernelErrchk();
	gpuErrchk(hipMemcpy(&noAs, d_out_numAs, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_chunk));
	gpuErrchk(hipFree(d_out_numAs));
	free(clearedChunk);
	if (noAs != DEBUG_A_COUNT)
	{
		throw std::runtime_error("invalid value calculated by tested function");
	}
}

template <int TNoBlocks>
void precleanedGPU(std::ifstream& fs)
{
	char* d_chunk;
	int* d_out_numAs;
	char* chunk = (char*)malloc(sizeof(char)*INPUT_CHUNK_SIZE);
	char* clearedChunk = (char*)malloc(sizeof(char)*INPUT_CHUNK_SIZE);
	int clearedChunkSize = 0;
	int chunkOffset = 0;
	int noAs = 0;
	int cutPhase = 2;
	gpuErrchk(hipMalloc(&d_chunk, INPUT_CHUNK_SIZE * sizeof(char)));
	gpuErrchk(hipMalloc(&d_out_numAs, sizeof(int)));
	gpuErrchk(hipMemset(&d_out_numAs, sizeof(int), 0));
	while (!fs.eof())
	{
		fs.read(chunk + chunkOffset, INPUT_CHUNK_SIZE - chunkOffset);
		int i = 0;
		for (int j = 0; j < 3 - cutPhase; j++)
		{
			while (chunk[i] != '\n')
			{
				i++;
			}
			i++;
		}
		bool endOfChunk = false;
		while (!endOfChunk)
		{
			int startOfLetters = i;
			while (chunk[i] != '\n')
			{
				i++;
				if (clearedChunkSize + i - startOfLetters == INPUT_CHUNK_SIZE)
				{
					gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
					AddPrecleanedChunkToGraph << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_out_numAs);
					kernelErrchk();
					clearedChunkSize = 0;
				}
				if (i == INPUT_CHUNK_SIZE)
				{
					memcpy(chunk, chunk + startOfLetters, sizeof(char)*(INPUT_CHUNK_SIZE - startOfLetters));
					chunkOffset = INPUT_CHUNK_SIZE - startOfLetters;
					cutPhase = 3;
					endOfChunk = true;
					break;
				}
			}
			i++;
			if (!endOfChunk)
			{
				memcpy(clearedChunk + clearedChunkSize, chunk + startOfLetters, i - startOfLetters);
				clearedChunkSize += i - startOfLetters;
			}
			if (!endOfChunk)
			{
				for (int j = 0; j < 3 && !endOfChunk; j++)
				{
					while (chunk[i] != '\n')
					{
						i++;
						if (i == INPUT_CHUNK_SIZE)
						{
							cutPhase = j;
							endOfChunk = true;
							break;
						}
					}
					i++;
				}
			}
		}
	}
	gpuErrchk(hipMemcpy(d_chunk, clearedChunk, clearedChunkSize * sizeof(char), hipMemcpyHostToDevice));
	AddPrecleanedChunkToGraph << <TNoBlocks, BLOCK_SIZE >> > (TNoBlocks, d_chunk, clearedChunkSize, d_out_numAs);
	kernelErrchk();
	gpuErrchk(hipMemcpy(&noAs, d_out_numAs, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_chunk));
	gpuErrchk(hipFree(d_out_numAs));
	free(chunk);
	free(clearedChunk);
	if (noAs != DEBUG_A_COUNT)
	{
		throw std::runtime_error("invalid value calculated by tested function");
	}
}

void simpleCPU(std::ifstream& fs)
{
	int noAs = 0;
	char* chunk = (char*)malloc(sizeof(char)*INPUT_CHUNK_SIZE);
	int cutPhase = 2;
	while (!fs.eof())
	{
		fs.read(chunk, INPUT_CHUNK_SIZE);
		int i = 0;
		for (int j=0; j < 3-cutPhase; j++)
		{
			while (chunk[i] != '\n')
			{
				i++;
			}
			i++;
		}
		bool endOfChunk = false;
		while (!endOfChunk)
		{
			int startOfLetters = i;
			while (chunk[i] != '\n')
			{
				if (chunk[i] == 'A')
				{
					noAs++;
				}
				i++;
				if (i == INPUT_CHUNK_SIZE)
				{
					//w jakis sposob musimy tu zapamietac to, co bedzie potrzebne do dalszego liczenia w kolejnym kawalku
					//moze to zapewne zajac troche czasu
					//ponizsze zakomentowane jest bledne
					//memcpy(chunk, chunk + startOfLetters, sizeof(char)*(INPUT_CHUNK_SIZE - startOfLetters));
					//chunkOffset = INPUT_CHUNK_SIZE - startOfLetters;
					cutPhase = 3;
					endOfChunk = true;
					break;
				}
			}
			i++;
			if (!endOfChunk)
			{
				for (int j = 0; j < 3 && !endOfChunk; j++)
				{
					while (chunk[i] != '\n')
					{
						i++;
						if (i == INPUT_CHUNK_SIZE)
						{
							cutPhase = j;
							endOfChunk = true;
							break;
						}
					}
					i++;
				}
			}
		}
	}
	free(chunk);
	if (noAs != DEBUG_A_COUNT)
	{
		throw std::runtime_error("invalid value calculated by tested function");
	}
}